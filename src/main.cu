#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <unordered_set>
#include <algorithm>
#include <chrono>
#include "tools.cu"
#include "triangulation.cu"
#include "findPaths.cu"

using namespace std::chrono;

#define THRESHOLD 25 // TODO WHICH VALUE?

// For a subproblem of n points, we put a limit of 10*n triangles generated.
// This is manually fixed because otherwise it would not have any limit and we have limited memory.
#define NB_MAX_TRIANGLES_PER_PT 10

//CPU Compare function
bool xCompare (float3 a, float3 b){return a.x < b.x;}

// Returns unique 64-bits int with 2 32-bits float
long unsigned int hash(float x, float y) {
    // Get binary representation of both numbers
    long unsigned int xInt = * (unsigned int *) &x;
    long unsigned int yInt = * (unsigned int *) &y;
    return (xInt << 32) + yInt;
}

std::vector<float3> readFile(std::string nameFile){
    std::vector<float3> pointsVector;

    std::ifstream inputFile;
    inputFile.open(nameFile);
    
    // Used to check if two points are identical
    std::unordered_set<long unsigned int> pointsSet;

    unsigned int i=0;
    float x,y;

    while(inputFile >> x >> y) {
        // Only push if the point is not over another
        long unsigned int hashValue = hash(x, y);
        if (!pointsSet.count(hashValue)) {
            pointsSet.insert(hashValue);
            pointsVector.push_back(make_float3(x, y, * (float *) &i));
            i++;
        }
    }

    std::cout << "Loaded file with " << pointsVector.size() << " distinct points\n" << std::endl;
    inputFile.close();

    return pointsVector;
    
}

void writeFile(int3* triangleList, int lineSize, int nbLines){
    std::ofstream indexFile("indexOutput.txt");

    for (int i = 0; i < nbLines; i++){
        for(int j = 0; j < lineSize; j++){
            if (triangleList[i*lineSize+j].x < 0)
                break;
            indexFile <<  triangleList[i*lineSize+j].x;
            indexFile <<  " ";
            indexFile <<  triangleList[i*lineSize+j].y;
            indexFile <<  " ";
            indexFile <<  triangleList[i*lineSize+j].z;
            indexFile << std::endl;
        }
    }
    indexFile.close();
}

int main(int argc, char *argv[]) {

    // arguments reading and errors (filename)
    if (argc < 2) {
        std::cout << "No input file provided" <<std::endl;
        return 1;
    }

    // -- Read original values
    std::vector<float3> pointsVector = readFile(argv[1]);
    int nbPoints = pointsVector.size();

    if (nbPoints < 1) {
        std::cout << "Input file is empty or does not exist." << std::endl;
        return 1;
    }

    // CPU Sorting values according to an axis
    std::sort(pointsVector.begin(), pointsVector.end(), xCompare);

    auto start = high_resolution_clock::now();

    // Copy points on GPU
    float3 *pointsOnGPU;
    long unsigned int mem = sizeof(float3) * nbPoints;
    hipMalloc((void**)&pointsOnGPU, mem);
    hipMemcpy(pointsOnGPU, &pointsVector[0], mem, hipMemcpyHostToDevice);

    // Find the number of subproblems according to the threshold of the
    // maximum number of points per subproblems. This number is always a power of 2

    // The way we have done our algorithm requires at least one split
    // To split the problem, the threshold is lowered if it is necesary
    int thresholdUsed = min(nbPoints-1, THRESHOLD);

    int nbSubproblems = 1, log2nbSubproblems = 0;
    while ((nbSubproblems * thresholdUsed) < nbPoints) {
        log2nbSubproblems++;
        nbSubproblems <<= 1;
    }

    struct edge* edgePathsList = createPaths(pointsOnGPU, nbPoints, nbSubproblems, log2nbSubproblems);

    int nbMaxTrianglesUsed = thresholdUsed * NB_MAX_TRIANGLES_PER_PT;

    int3* triangleList;
    int3* initTriangleList = new int3[nbSubproblems*nbMaxTrianglesUsed];

    struct edge* globalEdgeList;
    int boundMaxEdgePerSubset = (int)(2*nbPoints/nbSubproblems - 2)*3*3;
    edge* initGlobalEdgeList = new edge[boundMaxEdgePerSubset*nbSubproblems];

    for (int i = 0; i < boundMaxEdgePerSubset*nbSubproblems; i++){
        initGlobalEdgeList[i].usage = UNUSED;
    }

    hipMalloc((void**)&triangleList, sizeof(int3)*nbSubproblems*nbMaxTrianglesUsed); // FIXME Stored contiguously
    hipMemcpy(triangleList, initTriangleList, sizeof(int3)*nbSubproblems*nbMaxTrianglesUsed, hipMemcpyHostToDevice);

    hipMalloc((void**)&globalEdgeList, sizeof(edge)*boundMaxEdgePerSubset*nbSubproblems);
    hipMemcpy(globalEdgeList, initGlobalEdgeList, sizeof(edge)*boundMaxEdgePerSubset*nbSubproblems, hipMemcpyHostToDevice);

    std::cout << "Nb of subproblems: " << nbSubproblems << std::endl;

    parDeTri<<<nbSubproblems, 1>>>(pointsOnGPU, edgePathsList, globalEdgeList, triangleList, nbPoints, nbSubproblems, nbMaxTrianglesUsed);
    hipDeviceSynchronize();

    hipMemcpy(initTriangleList, triangleList, sizeof(int3)*nbSubproblems*nbMaxTrianglesUsed, hipMemcpyDeviceToHost);

    auto elapse = std::chrono::system_clock::now() - start;
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(elapse);
    std::cout << "Total duration : " << duration.count() << std::endl;

    writeFile(initTriangleList, nbMaxTrianglesUsed, nbSubproblems);

    hipFree(pointsOnGPU);
    hipFree(edgePathsList);
    hipFree(triangleList);
    hipFree(globalEdgeList);
    delete[] initTriangleList;
    delete[] initGlobalEdgeList;

    return 0;
}

// CPU time
// auto start = high_resolution_clock::now();
// auto elapse = std::chrono::system_clock::now() - start;
// auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(elapse);
// std::cout << duration.count() << std::endl;

// GPU time
// hipEvent_t myEvent, laterEvent;
// hipEventCreate(&myEvent);
// hipEventRecord(myEvent, 0);
// hipEventSynchronize(myEvent);
// int dimGrid = (nbPts+N-1)/N;   // Nb of blocks
// int dimBlock = N;
// projectPoints<<<dimGrid, dimBlock>>>(pointsOnGPU, pointsProjected, nbPts, true);
// hipDeviceSynchronize();

// hipEventCreate(&laterEvent);
// hipEventRecord(laterEvent, 0);
// hipEventSynchronize(laterEvent);