#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <unordered_set>
#include <algorithm>
#include <chrono>
#include "tools.cu"
#include "triangulation.cu"
#include "findPaths.cu"

using namespace std::chrono;

#define NB_MAX_TRIANGLES 10
#define THRESHOLD 5 // TODO WHICH VALUE?

//CPU Compare function
bool xCompare (float3 a, float3 b){return a.x < b.x;}
bool yCompare (float3 a, float3 b){return a.y < b.y;}

// Returns unique 64-bits int with 2 32-bits float
long unsigned int hash(float x, float y) {
    // Get binary representation of both numbers
    long unsigned int xInt = * (unsigned int *) &x;
    long unsigned int yInt = * (unsigned int *) &y;
    return (xInt << 32) + yInt;
}

std::vector<float3> readFile(std::string nameFile){
    std::vector<float3> pointsVector;

    std::ifstream inputFile;
    inputFile.open(nameFile);
    
    // Used to check if two points are identical
    std::unordered_set<long unsigned int> pointsSet;

    unsigned int i=0;
    float x,y;

    while(inputFile >> x >> y) {
        // Only push if the point is not over another
        long unsigned int hashValue = hash(x, y);
        if (!pointsSet.count(hashValue)) {
            pointsSet.insert(hashValue);
            pointsVector.push_back({x, y, * (float *) &i});
            i++;
        }
    }

    std::cout << "Loaded file with " << pointsVector.size() << " distinct points\n" << std::endl;
    inputFile.close();

    return pointsVector;
    
}

int main(int argc, char *argv[]) {

    // arguments reading and errors (filename)
    if (argc < 2) {
        std::cout << "No input file provided" <<std::endl;
        return 1;
    }

    // -- Read original values
    std::vector<float3> pointsVector = readFile(argv[1]);
    int nbPoints = pointsVector.size();

    std::cout << "Nb points : " << nbPoints << std::endl;

    // CPU Sorting values according to an axis
    std::sort(pointsVector.begin(), pointsVector.end(), xCompare);
    std::sort(pointsVector.begin(), pointsVector.end(), xCompare);

    float3 *pointsOnGPU;
    long unsigned int mem = sizeof(float3) * pointsVector.size();
    hipMalloc((void**)&pointsOnGPU, mem);
    hipMemcpy(pointsOnGPU, &pointsVector[0], mem, hipMemcpyHostToDevice);

    // Find the number of subproblems according to the threshold of the
    // maximum number of points per subproblems. This number is always a power of 2
    int nbSubproblems = 1, log2nbSubproblems = 0;
    while ((nbSubproblems * THRESHOLD) < nbPoints) {
        log2nbSubproblems++;
        nbSubproblems <<= 1;
    }

    struct edge* edgePathsList = createPaths(pointsOnGPU, nbPoints, nbSubproblems, log2nbSubproblems);

    int3* triangleList;
    int3 initTriangleList[nbSubproblems*NB_MAX_TRIANGLES];

    for (int i = 0; i < nbSubproblems; i++){
        for (int j = 0; j < NB_MAX_TRIANGLES; j++){
            initTriangleList[i*NB_MAX_TRIANGLES + j] = make_int3(-1, -1, -1);
        }
    }

    struct edge* globalEdgeList;
    int boundMaxEdgePerSubset = (int)(2*nbPoints/nbSubproblems - 2)*3*3;
    edge initGlobalEdgeList[boundMaxEdgePerSubset*nbSubproblems];

    for (int i = 0; i < boundMaxEdgePerSubset*nbSubproblems; i++){
        initGlobalEdgeList[i] = {make_float3(0, 0, 0), make_float3(0, 0, 0), UNUSED};
    }

    hipMalloc((void**)&triangleList, sizeof(int3)*nbSubproblems*NB_MAX_TRIANGLES); // FIXME Stored contiguously
    hipMemcpy(triangleList, initTriangleList, sizeof(int3)*nbSubproblems*NB_MAX_TRIANGLES, hipMemcpyHostToDevice);

    hipMalloc((void**)&globalEdgeList, sizeof(edge)*boundMaxEdgePerSubset*nbSubproblems);
    hipMemcpy(globalEdgeList, initGlobalEdgeList, sizeof(edge)*boundMaxEdgePerSubset*nbSubproblems, hipMemcpyHostToDevice);

    std::cout << "Nb of subproblems: " << nbSubproblems << std::endl;

    parDeTri<<<nbSubproblems, 1>>>(pointsOnGPU, edgePathsList, globalEdgeList, triangleList, nbPoints, nbSubproblems, NB_MAX_TRIANGLES);
    // hipDeviceSynchronize(); //TODO Required or not ?

    hipMemcpy(initTriangleList, triangleList, sizeof(int3)*nbSubproblems*NB_MAX_TRIANGLES, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < nbSubproblems*NB_MAX_TRIANGLES; i++){
        std::cout << "Triangle : " << initTriangleList[i].x << " " << initTriangleList[i].y << " " << initTriangleList[i].z << std::endl;
    }

    hipFree(edgePathsList);
    hipFree(pointsOnGPU);
    hipFree(globalEdgeList);

    return 0;
}

// CPU time
// auto start = high_resolution_clock::now();
// auto elapse = std::chrono::system_clock::now() - start;
// auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(elapse);