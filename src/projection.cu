#include "hip/hip_runtime.h"
#define N 1024 // TODO WHICH VALUE?

__global__ void projectPoint(point2D *pts, unsigned int refPointIndex, bool onAxisX) {
    point2D* localPoint = &pts[(blockIdx.x*N + threadIdx.x)];

    float old_x = localPoint->x;
    float old_y = localPoint->y;
    
    // TODO CHECK FASTER IF ARGUMENTS FOR BOTH VALUES
    float ref_x = pts[refPointIndex].x;
    float ref_y = pts[refPointIndex].y;

    float delta_y = ref_y - old_y;
    float delta_x = ref_x - old_x;

    // x takes delta between y values
    localPoint->x = onAxisX ? delta_y : delta_x;
    // y takes euclidian distance between points
    localPoint->y = delta_y * delta_y + delta_x * delta_x;
}

point2D* projection(std::vector<point2D> pointsVector) {
    point2D *res;

    long unsigned int mem = sizeof(point2D) * pointsVector.size();
    hipMalloc((void**)&res, mem);
    hipMemcpy(res, &pointsVector[0], mem, hipMemcpyHostToDevice);

    dim3 dimGrid((pointsVector.size()+N-1)/N, 1);   // Nb of blocks
    dim3 dimBlock(N, 1);
    projectPoint<<<dimGrid, dimBlock>>>(res, pointsVector.size()/2, true);

    hipDeviceSynchronize();

    point2D projection[pointsVector.size()]; // projection results
    hipMemcpy(projection, res, mem, hipMemcpyDeviceToHost);

    // for (int i=0; i<pointsVector.size(); i++) {
    //     std::cout << "Original: " << pointsVector[i].x << " " << pointsVector[i].y << std::endl;
    //     std::cout << "Projected: " << projection[i].x << " " << projection[i].y << std::endl;
    // }

    return res;
}