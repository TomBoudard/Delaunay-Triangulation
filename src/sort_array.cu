#include "hip/hip_runtime.h"
#include <stdio.h>
#include "point.cu"

__global__ void simple(point2D *r) {
    r[threadIdx.x].x = threadIdx.x;
}

// Sorts the vector into the GPU memory
// Returns pointer to sorted array in GPU
point2D* sortInputIntoGPU(std::vector<point2D> pointsVector) {
    point2D *res;
    int mem = sizeof(point2D) * pointsVector.size();
    hipMalloc((void**)&res, mem);

    hipMemcpy(res, &pointsVector, 10, hipMemcpyHostToDevice);

    dim3 dimGrid(mem, 1);
    dim3 dimBlock(sizeof(point2D), 1);
    simple<<<dimGrid, dimBlock>>>(res);

    hipMemcpy(&pointsVector, res, 10, hipMemcpyDeviceToHost);

    for (int i=0; i<10; i++) std::cout<< res[i].x <<" ";
    std::cout<<std::endl;

    return (point2D*) res;
}

