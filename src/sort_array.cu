#include "hip/hip_runtime.h"
#define NB_CORES 2
// Number of elements we should sort on one core (threshold)
#define NB_ELEM_SORT_CORE 5

#ifndef SORT_ARRAY
#define SORT_ARRAY

#include <stdio.h>
#include <cmath>
#include <vector>
#include <iostream>
#include "mesh.cu"

// Sorts a part of array
__global__ void sortGPU() {

}

// Sorts requested array stored on GPU. Updates input pointer
// Input:
//  - Pointer of the original array and length
// Output:
//  - Pointer of the sorted array
void sortArray(float3** input, unsigned int len) {

    // We do a merge sort so we will alternate between two buffers
    // The first one is the provided array, the other is allocated here
    float3* newBuffer;
    long unsigned int mem = sizeof(float3) * len;
    hipMalloc((void**)&newBuffer, mem);

    float3** buffers[2];
    buffers[0] = input;
    buffers[1] = &newBuffer;

    unsigned int currentInputBuffer = 0;

    // Input points the final buffer and the other one is free'd
    *input = *buffers[1-currentInputBuffer];
    hipFree(*buffers[currentInputBuffer]);

    return;
}


// __global__ void simple(float3 *r) {
//     r[threadIdx.x].x = threadIdx.x;
// }

// // Sorts the vector into the GPU memory
// // Returns pointer to sorted array in GPU
// float3* sortInputIntoGPU(std::vector<float3> pointsVector) {
//     float3 *res;
//     int mem = sizeof(float3) * pointsVector.size();
//     hipMalloc((void**)&res, mem);

//     hipMemcpy(res, &pointsVector, 10, hipMemcpyHostToDevice);

//     dim3 dimGrid(mem, 1);
//     dim3 dimBlock(sizeof(float3), 1);
//     simple<<<dimGrid, dimBlock>>>(res);

//     hipMemcpy(&pointsVector, res, 10, hipMemcpyDeviceToHost);

//     for (int i=0; i<10; i++) std::cout<< res[i].x <<" ";
//     std::cout<<std::endl;

//     return (float3*) res;
// }

#endif