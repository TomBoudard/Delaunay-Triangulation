#include "hip/hip_runtime.h"
#include "tools.cu"
#include <math_functions.h>

__device__ float delaunayDistance(float3 const& edgeStart, float3 const& edgeEnd, float3 const& point){

    float oppEdgeA = length(point-edgeEnd); 
    float oppEdgeB = length(edgeStart-point); 
    float oppEdgeC = length(edgeEnd-edgeStart);

    float barCircumCenterA = oppEdgeA*(oppEdgeB+oppEdgeC-oppEdgeA);
    float barCircumCenterB = oppEdgeB*(oppEdgeC+oppEdgeA-oppEdgeB);
    float barCircumCenterC = oppEdgeC*(oppEdgeA+oppEdgeB-oppEdgeC);

    float3 circumCenter;
    float sumBarycenters = barCircumCenterA + barCircumCenterB + barCircumCenterC;
    circumCenter.x = (barCircumCenterA*edgeStart.x + barCircumCenterB*edgeEnd.x + barCircumCenterC*point.x)/sumBarycenters;
    circumCenter.y = (barCircumCenterA*edgeStart.y + barCircumCenterB*edgeEnd.y + barCircumCenterC*point.y)/sumBarycenters;

    float radius = sqrt(length(circumCenter-point));

    if (barCircumCenterC < 0){ //Check whether the circumCenter is in the half space of the point or not
        return -radius;
    }
    else{
        return radius;
    }
}

__global__ void parDeTri(float3* points, edge* edgePathList, edge* globalEdgeList, int3* triangleList, int nbPoints, int nbSubproblems, int nbMaxTriangle){ //TODO Verify if possible to use reference //TODO Avoid if/else as much as possible
    
    printf("TEST START !\n");

    // IDs of the beginning and end of the slice manipulated by the block
    int sliceBlockBeg = (blockIdx.x) * nbPoints / (nbSubproblems);
    int sliceBlockEnd = ((blockIdx.x + 1)) * nbPoints / (nbSubproblems);

    int idLeft = ((int)log2((double)nbSubproblems) - __ffs(blockIdx.x) - 1)*nbPoints + blockIdx.x%((int)log2((double)nbSubproblems) - __ffs(blockIdx.x))*nbPoints/((int)log2((double)nbSubproblems) - __ffs(blockIdx.x));
    int idRight = ((int)log2((double)nbSubproblems) - __ffs(blockIdx.x+1) - 1)*nbPoints + (blockIdx.x+1)%((int)log2((double)nbSubproblems) - __ffs(blockIdx.x+1))*nbPoints/((int)log2((double)nbSubproblems) - __ffs(blockIdx.x+1));

    int copyIndex = 0;

    if (blockIdx.x != 0){
        for(int indexLeft = idLeft; indexLeft < nbPoints/((int)log2((double)nbSubproblems) - __ffs(blockIdx.x)); indexLeft++){
            globalEdgeList[copyIndex] = edgePathList[indexLeft];
            copyIndex++;
        }
    }
    if (blockIdx.x != (nbSubproblems - 1)){
        for(int indexRight = idRight; indexRight < nbPoints/((int)log2((double)nbSubproblems) - __ffs(blockIdx.x+1)); indexRight++){
            globalEdgeList[copyIndex] = edgePathList[indexRight];
            copyIndex++;
        }
    }

    int triangleIndex = (blockIdx.x)*nbMaxTriangle;

    int boundMaxEdgePerSubset = (int)(2*nbPoints/nbSubproblems - 2)*3*3;
    int startEdgeIndex = blockIdx.x*boundMaxEdgePerSubset*nbSubproblems;
    int endEdgeIndex = startEdgeIndex + copyIndex;

    edge currentEdge;
    float3 bestThirdPoint;
    int bestThirdPointSide;

    printf("TEST TRIANGULATION BEGIN !\n");

    //Triangulation
    while (startEdgeIndex < endEdgeIndex){

        currentEdge = globalEdgeList[startEdgeIndex];

        float bestRadius = INFINITY;
        bool triangleFound = false;
        for (int i = sliceBlockBeg; i<sliceBlockEnd; i++){
            float3 firstVector = currentEdge.y-currentEdge.x;
            float3 secondVector = points[i]-currentEdge.x;
            float zVectorialProduct = firstVector.x*secondVector.y - firstVector.y*secondVector.x;
            int pointSide = zVectorialProduct/fabs(zVectorialProduct);
            if(points[i].z != currentEdge.x.z && points[i].z != currentEdge.y.z && pointSide != currentEdge.usage && currentEdge.usage != FULL){ //If pointSide==currentEdge.z==0 it is skipped but we don't care about this case
                float radius = delaunayDistance(currentEdge.x, currentEdge.y, points[i]);
                
                if (radius < bestRadius){
                    int3 currentTriangle = make_int3(currentEdge.x.z, currentEdge.y.z, points[i].z);
                    bool alreadyExisting = false;
                    for (int j = blockIdx.x * nbMaxTriangle; j<triangleIndex; j++){
                        if (currentTriangle == triangleList[j]){
                            alreadyExisting = true;
                        }
                    }
                    if (!alreadyExisting){
                        bestRadius = radius;
                        bestThirdPoint = points[i];
                        bestThirdPointSide = pointSide;
                        triangleFound = true;
                    }
                }
            }
        }

        if (triangleFound){
            
            printf("TEST TRIANGLE FOUND : CHECK VALIDITY !\n");

            bool validTriangle = true;
            if (bestThirdPointSide == -1){//Means that the current edge is being used from y to x to be used in a direct repere
                float3 temp = currentEdge.x;
                currentEdge.x = currentEdge.y;
                currentEdge.y = temp;
            }
            edge secondEdge = {currentEdge.y, bestThirdPoint, 0};
            edge thirdEdge = {bestThirdPoint, currentEdge.x, 0};
            bool secondEdgeNew = true;
            bool thirdEdgeNew = true;
            int indexSecondEdge = -1; //TODO To optimize
            int indexThirdEdge = -1;
            for (int k = 0; k<=endEdgeIndex; k++){ // TODO To optimize if possible
                if (secondEdge == globalEdgeList[k]){
                    secondEdgeNew = false;
                    secondEdge = globalEdgeList[k];
                    indexSecondEdge = k;
                }
                if (thirdEdge == globalEdgeList[k]){
                    thirdEdgeNew = false;
                    thirdEdge = globalEdgeList[k];
                    indexThirdEdge = k;
                }
            }

            //The case where the edge has two points on the same side is not possible here otherwise the bestThirdPoint wouldn't have been choosen

            //Add the two side edges only if they are new (the first one is always preexisting)
            if (secondEdgeNew){
                secondEdge.usage = USED; //New edges are created according to a direct repere 
                globalEdgeList[endEdgeIndex] = secondEdge;
                endEdgeIndex++;
            }
            else{
                float3 firstVectorSecondEdge = secondEdge.y-secondEdge.x;
                float3 secondVectorSecondEdge = currentEdge.x-currentEdge.y;
                float zVectorialProductSecondEdge = firstVectorSecondEdge.x*secondVectorSecondEdge.y - firstVectorSecondEdge.y*secondVectorSecondEdge.x;
                if (zVectorialProductSecondEdge == 1 || secondEdge.usage == FULL){ //If used in the same way as created or already used twice
                    validTriangle = false;
                }
                if (secondEdge.usage == UNUSED){ // Special case happening only for the edges from the path
                    secondEdge.usage = USED;
                    globalEdgeList[indexThirdEdge].usage = secondEdge.usage;
                }
                else{
                    secondEdge.usage = FULL;
                    globalEdgeList[indexSecondEdge].usage = secondEdge.usage;
                }
            }

            if (thirdEdgeNew){
                thirdEdge.usage = USED; //New edges are created according to a direct repere 
                globalEdgeList[endEdgeIndex] = thirdEdge;
                endEdgeIndex++;
            }
            else{
                float3 firstVectorThirdEdge = thirdEdge.y-thirdEdge.x;
                float3 secondVectorThirdEdge = currentEdge.y-bestThirdPoint;
                float zVectorialProductThirdEdge = firstVectorThirdEdge.x*secondVectorThirdEdge.y - firstVectorThirdEdge.y*secondVectorThirdEdge.x;
                if (zVectorialProductThirdEdge == 1 || thirdEdge.usage == FULL){ //If they have the same sign
                    validTriangle = false;
                }
                if (thirdEdge.usage == UNUSED){ // Special case happening only for the edges from the path
                    thirdEdge.usage = USED;
                    globalEdgeList[indexThirdEdge].usage = thirdEdge.usage;
                }
                else{
                    thirdEdge.usage = FULL;
                    globalEdgeList[indexThirdEdge].usage = thirdEdge.usage;
                }
            }

            if(validTriangle){
                // std::cout << "THE TRIANGLE IS VALID" << std::endl;
                triangleList[triangleIndex].x = currentEdge.x.z;
                triangleList[triangleIndex].y = currentEdge.y.z;
                triangleList[triangleIndex].z = bestThirdPoint.z; //TODO STORE ONLY DIRECT TIRANGLE
                triangleIndex++;
                if (triangleIndex > (blockIdx.x+1)*nbMaxTriangle){
                    printf("/!\\ STOP : MAXIMUM AMOUNT OF TRIANGLE PER SUBSET EXCEEDED /!\\");
                }
            }
        }
        // std::cout << "======================" << std::endl;
        startEdgeIndex++;
    }
}