#include "hip/hip_runtime.h"
#include "tools.cu"
#include <hip/hip_runtime.h>

#define sub(a, b)make_float3(a.x - b.x, a.y - b.y, 0)

__device__ float delaunayDistance(float3 const& edgeStart, float3 const& edgeEnd, float3 const& point){

    float oppEdgeA = length(point-edgeEnd); 
    float oppEdgeB = length(edgeStart-point); 
    float oppEdgeC = length(edgeEnd-edgeStart);

    float barCircumCenterA = oppEdgeA*(oppEdgeB+oppEdgeC-oppEdgeA);
    float barCircumCenterB = oppEdgeB*(oppEdgeC+oppEdgeA-oppEdgeB);
    float barCircumCenterC = oppEdgeC*(oppEdgeA+oppEdgeB-oppEdgeC);

    float3 circumCenter;
    float sumBarycenters = barCircumCenterA + barCircumCenterB + barCircumCenterC;
    circumCenter.x = (barCircumCenterA*edgeStart.x + barCircumCenterB*edgeEnd.x + barCircumCenterC*point.x)/sumBarycenters;
    circumCenter.y = (barCircumCenterA*edgeStart.y + barCircumCenterB*edgeEnd.y + barCircumCenterC*point.y)/sumBarycenters;

    float radius = sqrt(length(circumCenter-point));

    if (barCircumCenterC < 0){ //Check whether the circumCenter is in the half space of the point or not
        return -radius;
    }
    else{
        return radius;
    }
}

__global__ void parDeTri(float3* points, edge* edgePathList, edge* globalEdgeList, int3* triangleList, int nbPoints, int nbSubproblems, int nbMaxTriangle){
    
    // IDs of the beginning and end of the slice manipulated by the block
    int sliceBlockBeg = (blockIdx.x) * nbPoints / (nbSubproblems);
    int sliceBlockEnd = ((blockIdx.x + 1)) * nbPoints / (nbSubproblems);

    float mostLeftX = points[sliceBlockBeg].x;
    float mostRightX = points[sliceBlockEnd].x;

    int nbLeftNeighbours = 0;
    int nbRightNeighbours = 0;

    int log2nbSubproblems = __ffs(nbSubproblems) - 1;

    // Compute the ids of the (at most) two path(s)
    int idLeftRow = (log2nbSubproblems - (__ffs(blockIdx.x)));
    int idLeftCol = (blockIdx.x - (1 << (log2nbSubproblems - idLeftRow-1))) / (1 << (log2nbSubproblems - idLeftRow)) * nbPoints / (1 << idLeftRow);
    int idRightRow = (log2nbSubproblems - (__ffs(blockIdx.x + 1)));
    int idRightCol = (blockIdx.x+1 - (1 << (log2nbSubproblems - idRightRow-1))) / (1 << (log2nbSubproblems - idRightRow)) * nbPoints / (1 << idRightRow);

    int idLeft = idLeftRow * nbPoints + idLeftCol;
    int idRight = idRightRow * nbPoints + idRightCol;

    int boundMaxEdgePerSubset = (int)(2*nbPoints/nbSubproblems - 2)*3*3;
    int copyIndex = blockIdx.x*boundMaxEdgePerSubset*nbSubproblems;

    if (blockIdx.x != 0){
        while (edgePathList[idLeft].usage != INVALID){
            globalEdgeList[copyIndex] = edgePathList[idLeft];
            globalEdgeList[copyIndex].usage = UNUSED_LEFT;
            idLeft++;
            copyIndex++;
        }
    }
    if (blockIdx.x != (nbSubproblems - 1)){
        while (edgePathList[idRight].usage != INVALID){
            globalEdgeList[copyIndex] = edgePathList[idRight];
            globalEdgeList[copyIndex].usage = UNUSED_RIGHT;
            idRight++;
            copyIndex++;
        }
    }

    int triangleIndex = (blockIdx.x)*nbMaxTriangle;

    int initialStartEdgeIndex = blockIdx.x*boundMaxEdgePerSubset*nbSubproblems;
    int startEdgeIndex = initialStartEdgeIndex;
    int endEdgeIndex = copyIndex;

    edge currentEdge;
    float3 bestThirdPoint;
    int bestThirdPointSide;

    //Triangulation
    while (startEdgeIndex < endEdgeIndex){ //Sliding window

        currentEdge = globalEdgeList[startEdgeIndex];

        float bestRadius = INFINITY;
        bool triangleFound = false;
        for (int i = sliceBlockBeg - nbLeftNeighbours; i<sliceBlockEnd + nbRightNeighbours; i++){
            float3 firstVector = currentEdge.y - currentEdge.x;
            float3 secondVector = points[i] - currentEdge.x;
            float zVectorialProduct = firstVector.x*secondVector.y - firstVector.y*secondVector.x;
            int pointSide = 0;
            if (zVectorialProduct > 0){
                pointSide = 1;
            }
            else if(zVectorialProduct < 0){
                pointSide = -1;
            }

            if(points[i].z != currentEdge.x.z && points[i].z != currentEdge.y.z && pointSide != 0
                && pointSide != currentEdge.usage && currentEdge.usage != FULL
                && !(currentEdge.usage == UNUSED_LEFT && pointSide == 1)
                && !(currentEdge.usage == UNUSED_RIGHT && pointSide == -1)){
                    
                float radius = delaunayDistance(currentEdge.x, currentEdge.y, points[i]);
                if (radius < bestRadius){
                    int3 currentTriangle = make_int3(currentEdge.x.z, currentEdge.y.z, points[i].z);
                    bool alreadyExisting = false;
                    for (int j = blockIdx.x * nbMaxTriangle; j<triangleIndex; j++){
                        if (currentTriangle == triangleList[j]){
                            alreadyExisting = true;
                        }
                    }
                    if (!alreadyExisting){
                        bestRadius = radius;
                        bestThirdPoint = points[i];
                        bestThirdPointSide = pointSide;
                        triangleFound = true;
                    }
                }
            }
        }

        if (triangleFound){
            
            bool validTriangle = true;
            if (bestThirdPointSide == -1){ //Means that the current edge must be used from y to x to be used in a direct repere
                float3 temp = currentEdge.x;
                currentEdge.x = currentEdge.y;
                currentEdge.y = temp;
            }
            edge secondEdge = {currentEdge.y, bestThirdPoint, 0};
            edge thirdEdge = {bestThirdPoint, currentEdge.x, 0};
            bool secondEdgeNew = true;
            bool thirdEdgeNew = true;
            int indexSecondEdge = -1;
            int indexThirdEdge = -1;
            for (int k = initialStartEdgeIndex; k<=endEdgeIndex; k++){
                if (secondEdge == globalEdgeList[k]){
                    secondEdgeNew = false;
                    secondEdge = globalEdgeList[k];
                    indexSecondEdge = k;
                }
                if (thirdEdge == globalEdgeList[k]){
                    thirdEdgeNew = false;
                    thirdEdge = globalEdgeList[k];
                    indexThirdEdge = k;
                }
            }

            //Add the two side edges only if they are new (the first one is always preexisting)
            if (secondEdgeNew){
                secondEdge.usage = USED; //New edges are created according to a direct repere 
                globalEdgeList[endEdgeIndex] = secondEdge;
                endEdgeIndex++;
            }
            else{
                float3 firstVectorSecondEdge = secondEdge.y-secondEdge.x;
                float3 secondVectorSecondEdge = currentEdge.x-currentEdge.y;
                float zVectorialProductSecondEdge = firstVectorSecondEdge.x*secondVectorSecondEdge.y - firstVectorSecondEdge.y*secondVectorSecondEdge.x;
                if (zVectorialProductSecondEdge == 1 || secondEdge.usage == FULL){ //If used in the same way as created or already used twice
                    validTriangle = false;
                }
                else{
                    secondEdge.usage = FULL; // The use of a preexisting edge necessarily make it used twice (on both sides) or an edge of the path can be used only once in a subproblem
                    globalEdgeList[indexSecondEdge].usage = secondEdge.usage;
                }
            }

            if (thirdEdgeNew){
                thirdEdge.usage = USED; //New edges are created according to a direct repere 
                globalEdgeList[endEdgeIndex] = thirdEdge;
                endEdgeIndex++;
            }
            else{
                float3 firstVectorThirdEdge = thirdEdge.y-thirdEdge.x;
                float3 secondVectorThirdEdge = currentEdge.y-bestThirdPoint;
                float zVectorialProductThirdEdge = firstVectorThirdEdge.x*secondVectorThirdEdge.y - firstVectorThirdEdge.y*secondVectorThirdEdge.x;
                if (zVectorialProductThirdEdge == 1 || thirdEdge.usage == FULL){ //If used in the same way as created or already used twice
                    validTriangle = false;
                }
                else{
                    thirdEdge.usage = FULL;  // The use of a preexisting edge necessarily make it used twice (on both sides) or an edge of the path can be used only once in a subproblem
                    globalEdgeList[indexThirdEdge].usage = thirdEdge.usage;
                }
            }

            if(validTriangle){
                currentEdge.usage = FULL; // The use of a preexisting edge necessarily make it used twice (on both sides) or an edge of the path can be used only once in a subproblem
                globalEdgeList[startEdgeIndex].usage = currentEdge.usage;

                triangleList[triangleIndex].x = *(int*)& currentEdge.x.z;
                triangleList[triangleIndex].y = *(int*)& currentEdge.y.z;
                triangleList[triangleIndex].z = *(int*)& bestThirdPoint.z;

                triangleIndex++;
                triangleList[triangleIndex].x = -1;

                if (triangleIndex > (blockIdx.x+1)*nbMaxTriangle){
                    printf("/!\\ STOP : MAXIMUM AMOUNT OF TRIANGLE PER SUBSET EXCEEDED /!\\ \n");
                }
            }
        }
        startEdgeIndex++;
    }
}